#include "hip/hip_runtime.h"
#define TILE_WIDTH 32
#define COARSE_FACTOR 4

__global__ void matrixMulKernel(float *M, float *N, float *p, int width) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and colums of the P element to work on
  int row = by * TILE_WIDTH + ty;
  int colStart = bx * TILE_WIDTH * COARSE_FACTOR + tx;

  // Initialize Pvalue for all output elements
  float Pvalue[COARSE_FACTOR];
  for (int c = 0; c < COARSE_FACTOR; ++c) {
    Pvalue[c] = 0.0f;
  }

  // Loop over the M and N tiles required to compute P element
  for (int ph = 0; ph < width / TILE_WIDTH; ++ph) {
    // Collaborative loading of M tile into shared memory
    Mds[ty][tx] = M[row * width + ph * TILE_WIDTH + tx];

    for (int c = 0; c < COARSE_FACTOR; ++c) {
      int col = colStart + c * TILE_WIDTH;

      // Collaborative loading of N tile into shared memory
      Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * width + col];
      __syncthreads();

      for (int k = 0; k < TILE_WIDTH; ++k) {
        Pvalue[c] += Mds[ty][k] * Nds[k][tx];
      }
      __syncthreads();
    }
  }

  for (int c = 0; c < COARSE_FACTOR; ++c) {
    int col = colStart + c * TILE_WIDTH;
    P[row * width + col] = Pvalue[c];
  }
}