#include "hip/hip_runtime.h"
__global__ void histo_kernel(char *data, unsigned int length,
                             unsigned int *histo) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < length) {
    int alphabet_position = data[i] - 'a';
    if (alphabet_position >= 0 && alphabet_position < 26) {
      atomicAdd(&(histo[alphabet_position / 4]), 1);
    }
  }
}

__global__ void histo_private_kernel(char *data, unsigned int length,
                                     unsigned int *histo) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < length) {
    int alphabet_position = data[i] - 'a';
    if (alphabet_position >= 0 && alphabet_position < 26) {
      atomicAdd(&(histo[blockIdx.x * NUM_BINS + alphabet_position / 4]), 1);
    }
  }

  if (blockIdx.x > 0) {
    __synchthreads();
    for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
      unsigned int binValue = histo[blockIdx.x * NUM_BINS + bin];
      if (binValue > 0) {
        atomicAdd(&histo[bin], binValue);
      }
    }
  }
}

__global__ void histo_shared_memory_private_kernel(char8 data,
                                                   unsigned int length,
                                                   unsigned int *histo) {
  // Initialize privatized bins
  __shared__ unsigned int histo_s[NUM_BINS];
  for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
    histo_s[bin] = 0;
  }
  __synchthreads();

  // Histogram
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < length) {
    int alphabet_position = data[i] - 'a';
    if (alphabet_position >= 0 && alphabet_position < 26) {
      atomicAdd(&(histo_s[alphabet_position / 4]), 1);
    }
  }
  __synchthreads();

  for (unsigned int bin = threadIdx.x; bin < NUM_BINS; bin += blockDim.x) {
    unsigned int binValue = histo_s[bin];
    if (binValue > 0) {
      atomicAdd(&(histo[bin]), binValue);
    }
  }
}